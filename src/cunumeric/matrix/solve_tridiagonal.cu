/* Copyright 2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/matrix/solve_tridiagonal.h"
#include "cunumeric/matrix/solve_tridiagonal_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace legate;

template <typename Getgtsv2BufferSize, typename Getgtsv2Solver, typename VAL>
static inline void solve_new_template(Getgtsv2BufferSize gtsv2_buffer_size, 
                                  Getgtsv2Solver gtsv2solver,
                                  int32_t m,
                                  int32_t n,
                                  VAL *dl,
                                  VAL *d,
                                  VAL *du, 
                                  VAL *B,
                                  int32_t ldb)
{
  auto handle = get_cusparse(); 
  auto stream = get_cached_stream();
  CHECK_CUSPARSE(hipsparseSetStream(handle, stream));

  size_t buffer_size;

  CHECK_CUSPARSE(gtsv2_buffer_size(handle, m, n, dl, d, du, B, ldb, &buffer_size));

#if DEBUG_CUNUMERIC
  assert(buffer_size % 128 == 0);
#endif

  // std::cout << "buffer size is: " << buffer_size << std::endl;
  // std::cout <<"m, n, ldb: " << m << " " << n << " " << ldb << std::endl;

  // Allocate the buffer
  void* buffer = nullptr;
  if (buffer_size > 0) {
    Legion::DeferredBuffer<char, 1> buf({0, buffer_size - 1}, Memory::GPU_FB_MEM);
    buffer = (void *) buf.ptr(0);
  } 
  
  CHECK_CUSPARSE(gtsv2solver(handle, m, n, dl, d, du, B, ldb, reinterpret_cast<void *>(buffer)));
  CHECK_CUDA(hipStreamSynchronize(stream));

}

template<>
struct SolveTridiagonalImplBody<VariantKind::GPU, Type::Code::FLOAT32> {
  void operator()(int32_t m, int32_t n, float* dl, float* d, float* du, float* B, int32_t ldb)
  {
    solve_new_template(hipsparseSgtsv2_bufferSizeExt,
                       hipsparseSgtsv2,
                       m, 
                       n, 
                       dl, 
                       d, 
                       du, 
                       B, 
                       ldb);
  }
};

template<>
struct SolveTridiagonalImplBody<VariantKind::GPU, Type::Code::FLOAT64> {
  void operator()(int32_t m, int32_t n, double* dl, double* d, double* du, double* B, int32_t ldb)
  {
    solve_new_template(hipsparseDgtsv2_bufferSizeExt,
                       hipsparseDgtsv2,
                       m, 
                       n, 
                       dl, 
                       d, 
                       du, 
                       B, 
                       ldb);
  }
};

/*

template<>
struct SolveTridiagonalImplBody<VariantKind::GPU, Type::Code::COMPLEX64> {
  void operator()(int32_t m, int32_t n, complex<float>* dl, complex<float>* d, complex<float>* du, complex<float>* B, int32_t ldb)
  {
    solve_new_template(hipsparseCgtsv2_bufferSizeExt,
                       hipsparseCgtsv2,
                       m, 
                       n, 
                       reinterpret_cast<hipComplex*>(dl),
                       reinterpret_cast<hipComplex*>(d),
                       reinterpret_cast<hipComplex*>(du),
                       reinterpret_cast<hipComplex*>(B),
                       ldb);

  }
};

template<>
struct SolveTridiagonalImplBody<VariantKind::GPU, Type::Code::COMPLEX128> {
  void operator()(int32_t m, int32_t n, complex<double>* dl, complex<double>* d, complex<double>* du, complex<double>* B, int32_t ldb)
  {
    solve_new_template(hipsparseZgtsv2_bufferSizeExt,
                       hipsparseZgtsv2,
                       m, 
                       n, 
                       reinterpret_cast<hipDoubleComplex*>(dl),
                       reinterpret_cast<hipDoubleComplex*>(d),
                       reinterpret_cast<hipDoubleComplex*>(du),
                       reinterpret_cast<hipDoubleComplex*>(B),
                       ldb);
  }
};

*/

/*static*/ void SolveTridiagonalTask::gpu_variant(TaskContext& context)
{
  solve_tridiagonal_template<VariantKind::GPU>(context);
}

} // namespace cunumeric


